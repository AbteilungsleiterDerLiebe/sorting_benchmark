/*
http://stackoverflow.com/questions/26206544/parallel-radix-sort-how-would-this-implementation-actually-work-are-there-some
*/

#include "cuda_radix_sort.cuh"
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>

cuda_radix_sort::cuda_radix_sort()
{
}

void cuda_radix_sort::run(int * unsortedInts, int length)
{
	hipSetDevice(0);
	int* v;
	hipMalloc(&v, length * sizeof(int));
	hipMemcpy(v, unsortedInts, length * sizeof(int), hipMemcpyHostToDevice);

	try {
		thrust::sort(thrust::device_ptr<int>(v), thrust::device_ptr<int>(v + length));
	}
	catch (thrust::system_error &e) {
		printf("Error: %s \n", e.what());
	}

	//// generate 32M random numbers serially

	//thrust::host_vector<int> h_vec(unsortedInts, unsortedInts + length);

	//// transfer data to the device

	//thrust::device_vector<int> d_vec = h_vec;

	//// sort data on the device (Only HOST VECTOR compiles!)

	//thrust::sort(d_vec.begin(), d_vec.end());

	//// transfer data back to host

	//thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
}



void cuda_radix_sort::radix_sort(int * values)
{

}