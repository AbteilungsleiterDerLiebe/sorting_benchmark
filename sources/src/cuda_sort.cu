#include "hip/hip_runtime.h"
#include "cuda_sort.cuh"
#include <algorithm>

cuda_sort::cuda_sort()
{
	name = "cuda Bitonic sort";
}

__global__
void bitonic_sort_step(int * dev_values, int j, int k)
{
	// sorting params
	unsigned int i, ixj;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	ixj = i^j;

	// threads with the lowest ids sort the array
	if ((ixj) > i) {
		if ((i&k) == 0) {
			// sort ascending
			if (dev_values[i] > dev_values[ixj]) {
				// swap i with ixj
				int temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
		if ((i&k) != 0) {
			// sort descending
			if (dev_values[i] < dev_values[ixj]) {
				// swap ixj with i
				int temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
	}
}


void cuda_sort::run(int * unsortedInts, int length)
{
	THREADS = 512;
	NUM_VALS = length;
	BLOCKS = NUM_VALS / THREADS;

	int *dev_values;
	size_t size = NUM_VALS * sizeof(int);

	hipMalloc((void**)&dev_values, size);
	hipMemcpy(dev_values, unsortedInts, size, hipMemcpyHostToDevice);

	dim3 blocks(BLOCKS, 1); // num blocks
	dim3 threads(THREADS, 1); // num threads

	int j, k;

	// Major step
	for (k = 2; k <= NUM_VALS; k <<= 1) {
		// Minor step
		for (j = k >> 1; j > 0; j = j >> 1) {
			bitonic_sort_step <<<blocks, threads >>>(dev_values, j, k);
		}
	}
	int * res;
	hipMemcpy(unsortedInts, dev_values, size, hipMemcpyDeviceToHost);
	hipFree(dev_values);
}


void cuda_sort::bitonic_sort(int * values)
{
	std::vector<int> test(values, values + NUM_VALS);
	int *dev_values;
	size_t size = NUM_VALS * sizeof(int);

	hipMalloc((void**)&dev_values, size);
	hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

	dim3 blocks(BLOCKS, 1); // num blocks
	dim3 threads(THREADS, 1); // num threads

	int j, k;

	// Major step
	for (k = 2; k <= NUM_VALS; k <<= 1) {
		// Minor step
		for (j = k >> 1; j > 0; j = j >> 1) {
			bitonic_sort_step <<<blocks, threads>>>(dev_values, j, k);
		}
	}

	hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
	hipFree(dev_values);
}