#include "hip/hip_runtime.h"
#include "cuda_sort.cuh"
#include <iostream>
#include <vector>
#include <time.h>
#include <algorithm>

// defines
#define THREADS 512 // 2^9
#define BLOCKS 131072 // 2^15
#define NUM_VALS THREADS*BLOCKS

void print_elapsed(clock_t start, clock_t stop)
{
	double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
	printf("Elapsed time: %.3fs\n", elapsed);
}

int random_int()
{
	return (int)rand();
}

void array_fill(int *arr, int length)
{
	int i;
	for (i = 0; i < length; ++i) {
		arr[i] = random_int();
	}
}

int cuda_sort::test_output()
{

	clock_t start, stop;

	

	int * values = (int*)malloc(NUM_VALS * sizeof(int));
	array_fill(values, NUM_VALS);

	// sort this with std lib
	std::vector<int> value_vec(values, values + NUM_VALS);
	std::vector<int> value_vec2(values, values + NUM_VALS);
	
	start = clock();
	bitonic_sort(values);
	stop = clock();

	std::cout << "Number of elements: " << value_vec2.size() << std::endl;
	print_elapsed(start, stop);

	start = clock();
	std::sort(value_vec2.begin(), value_vec2.end());
	stop = clock();


	std::cout << "same array with std::sort: " << std::endl;
	print_elapsed(start, stop);
	/*	
	for each (int var in value_vec2)
	{
		std::cout << var << std::endl;
	}
	*/
	


	return 42;
}

__global__
void bitonic_sort_step(int * dev_values, int j, int k)
{
	// sorting params
	unsigned int i, ixj;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	ixj = i^j;
	
	// threads with the lowest ids sort the array
	if ((ixj) > i) {
		if ((i&k) == 0) {
			// sort ascending
			if (dev_values[i] > dev_values[ixj]) {
				// swap i with ixj
				int temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
		if ((i&k) != 0) {
			// sort descending
			if (dev_values[i] < dev_values[ixj]) {
				// swap ixj with i
				int temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
	}
}

void cuda_sort::bitonic_sort(int * values)
{
	int *dev_values;
	size_t size = NUM_VALS * sizeof(int);

	hipMalloc((void**)&dev_values, size);
	hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

	dim3 blocks(BLOCKS, 1); // num blocks
	dim3 threads(THREADS, 1); // num threads

	int j, k;

	// Major step
	for (k = 2; k <= NUM_VALS; k <<= 1) {
		// Minor step
		for (j = k >> 1; j > 0; j = j >> 1) {
			bitonic_sort_step <<<blocks, threads>>>(dev_values, j, k);
		}
	}
	hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
	hipFree(dev_values);
}
